#include <hip/hip_runtime.h>

__global__ void convolution_1d_kernel(const float* input, const float* kernel, float* output,
                                      int input_size, int kernel_size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < input_size - kernel_size + 1) {
        output[idx] = 0.0f;
        for (int i = 0; i < kernel_size; i++) {
            output[idx] += input[idx+i] * kernel[i];
        }
    }
}

// input, kernel, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* input, const float* kernel, float* output, int input_size, int kernel_size) {
    int output_size = input_size - kernel_size + 1;
    int threadsPerBlock = 256;
    int blocksPerGrid = (output_size + threadsPerBlock - 1) / threadsPerBlock;

    convolution_1d_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, kernel, output, input_size, kernel_size);
    hipDeviceSynchronize();
}
