#include <hip/hip_runtime.h>

__global__ void invert_kernel(unsigned char* image, int width, int height) {
    int col = (blockIdx.x * blockDim.x + threadIdx.x) * 4;

    if (col < width * height * 4) {
        for (int i = 0; i < 3; i++) {
            image[col + i] = 255 - image[col + i];
        }
    }
}

// image_input, image_output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(unsigned char* image, int width, int height) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;

    invert_kernel<<<blocksPerGrid, threadsPerBlock>>>(image, width, height);
    hipDeviceSynchronize();
}
