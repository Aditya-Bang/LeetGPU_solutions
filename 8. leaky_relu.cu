#include <hip/hip_runtime.h>

__global__ void leaky_relu_kernel(const float* input, float* output, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    float ALPHA = 0.01;
    float ins = input[idx];

    if (idx < N) {
        output[idx] = ins * (ins > 0) + ALPHA * ins * (ins <= 0); // try to reduce warp divergence
    }
}

// input, output are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* input, float* output, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    
    leaky_relu_kernel<<<blocksPerGrid, threadsPerBlock>>>(input, output, N);
    hipDeviceSynchronize();
}
