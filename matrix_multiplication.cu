#include <hip/hip_runtime.h>

__global__ void matrix_multiplication_kernel(const float* A, const float* B, float* C, int M, int N, int K) {
    int j = blockIdx.x * blockDim.x + threadIdx.x;
    int i = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < M && j < K) {
        float sum = 0.0f;
        for (int k = 0; k < N; k++) {
            sum += A[i * N + k] * B[k * K + j];
        }
        C[i * K + j] = sum;
    }
}

// A, B, C are device pointers (i.e. pointers to memory on the GPU)
extern "C" void solve(const float* A, const float* B, float* C, int M, int N, int K) {
    dim3 threadsPerBlock(16, 16);
    dim3 blocksPerGrid((K + threadsPerBlock.x - 1) / threadsPerBlock.x,
                       (M + threadsPerBlock.y - 1) / threadsPerBlock.y);
    
    matrix_multiplication_kernel<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, M, N, K);
    hipDeviceSynchronize();
}
