#include <hip/hip_runtime.h>

__global__ void reverse_array(float* input, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < N - 1 - idx) { // no race condition in editing v.s. reading array because of this condition
        float tmp = input[idx];
        input[idx] = input[N - 1 - idx];
        input[N - 1 - idx] = tmp; 
    }
}

// input is device pointer
extern "C" void solve(float* input, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    reverse_array<<<blocksPerGrid, threadsPerBlock>>>(input, N);
    hipDeviceSynchronize();
}
